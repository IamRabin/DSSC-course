#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

const int N=8192;  //matrix size of N*N
const int k=8;     //tile size is  N*N
const int repetiton=100; // number of repetiton to avearge out the elapsed time .



int compare_matrices(float *gpu_mat, float *ref_mat)
{
	int result = 0;

	for(int j=0; j < N; j++)
    for(int i=0; i < N; i++)
    		if (ref_mat[i + j*N] != gpu_mat[i + j*N])
    		{
    			result = 1;
    		}
    return result;
}

/**
 * [make_matrix function fills the matrix with numbers from 0 to N-1]
 */

void  make_matrix(float * mat)
{
    for (int j=0; j<N*N; j++)
    {
        mat[j]=(float) j;
    }

}

/**
 * [prints matrix ]
 */

void print_matrix(float *mat)
{
	for(int j=0; j < N; j++)
	{
		for(int i=0; i < N; i++) { printf("%4.4g ", mat[i + j*N]); }
		printf("\n");
	}
}


/**
 * [transpose_CPU with row major order]
 *
 */

void
transpose_CPU(float in [ ], float out [ ] )
{
	for(int j=0; j < N; j++)
    for(int i=0; i < N; i++)
      		out [j + i*N] = in [i + j*N]; // reading and writing in row major order,  out(j,i) = in(i,j)
}

// ------------------------
 // Naive Transpose
// -------------------------

// Simple naive transpose which does not use shared memory.
// to be launched with one thread per element, in KxK threadblocks
// thread (x,y) in grid writes element (i,j) of output matrix
__global__ void
transpose_naive(float in[], float out[])
{
	int i = blockIdx.x * k + threadIdx.x;
	int j = blockIdx.y  * k + threadIdx.y;

	out[j + i*N] = in[i + j*N]; // reading is coaelsced but writing is uncoaelsced.
}


// ------------------------
 // transposeCoalesced
// -------------------------

// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elements
//Threads in  a thread block will copy the tile into shared memory and perform the transpose in the
//the shared memory which will be finally copied to global memory.

__global__ void
transpose_parallel_per_element_tiled(float in [],float out[])
{
    // (i,j) locations of the tile corner for input & output matrices:This shows where we start reading
    // from input and where to start writing in the output.

    int  in_corner_i= blockIdx.x * k,  in_corner_j=blockIdx.y * k; //which out(j,i) = in(i,j) block we are in mulltiplied by width of the tile.
    int out_corner_i=blockIdx.y * k,  out_corner_j = blockIdx.x * k; //output simply inverts y and x

    int x= threadIdx.x, y=threadIdx.y; //assigning x and y to thread index, tells which elemnt to read and write.

    __shared__  float tile[k][k]; //declare floating point array in shared memory for the tile

// coalesced read from global mem, TRANSPOSED write into shared mem:
tile [y][x] = in [(in_corner_i +x )+(in_corner_j+y)*N];// where tile starts + which element the particular thread is responsible for.

__syncthreads(); //threads wait here at the barrier

//read from shared mem,coalesced write to global mem:
out[(out_corner_i+x)+(out_corner_j+y)*N]=tile[x][y];//adjacent threads are varying by x

}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////


int main(int argc, char **argv)
{
    int numbytes= N*N* sizeof (float); // size of the matrix

   // allocate for input and output matrices
    float * in=(float*) malloc(numbytes);
    float * out=(float*)malloc (numbytes);
    float * gold_standard=(float*) malloc(numbytes);

   //calling the routine fill_matrix which inserts sequential numbers into the matrix.
    make_matrix(in);
    transpose_CPU(in, gold_standard); // transpose routine which puts result in  gold_standard for ref. comaprison.

   // allocate space for input and output matrix on device
    float *d_in,  *d_out ;

    hipMalloc(&d_in, numbytes);
    hipMalloc(&d_out, numbytes);
    hipMemcpy(d_in,in,numbytes,hipMemcpyHostToDevice); //copy matrix to the device
    GpuTimer timer;

	dim3 blocks(N/k,N/k); // blocks per grid
    dim3 threads(k,k); // threads per block


    timer.Start();
	 for (int i=0; i<repetiton;i++)
	 {
	         transpose_naive<<<blocks,threads>>>(d_in, d_out); //launching kernel running in single thread in single block.
      }
	timer.Stop();
	float avg_time=timer.Elapsed()/100;
    float bandwith= N*N*4*2*1e-6/avg_time;


	//copying the matrix from device to host
 	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);

 	printf("transpose_naive: %g ms.\nVerifying transpose...%s\n",
    avg_time, compare_matrices(out, gold_standard) ? "Unsuccessful" : "Successful");
    printf("Bandwidth: %g GB/s.\n",bandwith );
	printf(" *********************************** \n ");


    timer.Start();
	 for (int i=0; i<repetiton;i++)
	 {
	     transpose_parallel_per_element_tiled<<<blocks,threads>>>(d_in, d_out);
	 }
	timer.Stop();
	float avg_time2=timer.Elapsed()/100;
    float Bandwith= N*N*4*2*1e-6/avg_time2;

    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);

	printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n",
    k, k, avg_time2, compare_matrices(out, gold_standard) ? "Unsuccessful" : "Successful");
    printf("Bandwidth: %g GB/s.\n",Bandwith);


    hipFree(d_in);
    hipFree(d_out);
}
